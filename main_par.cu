#include "hip/hip_runtime.h"
#include "iter.cuh"
#include "mesh.h"
#include <mpi.h>
#include <getopt.h>
#include <cstring>
#include <fstream>
#include <cstring>
#include <cstdlib>
#include <iostream>
#include <cmath>

double phi(PointD p){
    double x = p.first;
    double y = p.second;
    return sqrt(4 + x*y);
}
PointUi splitFunction(int N0, int N1, int p) {
    double n0, n1;
    int p0, i;

    n0 = (double) N0; n1 = (double) N1;
    p0 = 0;

    for(i = 0; i < p; i++) {
        if(n0 > n1) {
            n0 = n0 / 2.0;
            ++p0;
        } else {
            n1 = n1 / 2.0;
        }
    }
    return PointUi(p0, p-p0);
}

int getPowOfTwo(int val){
    int pwr = 0;
    while(val >>= 1) ++pwr;
    return pwr;
}

struct Params {
    long a;
    long b;
    long rows;
    long cols;
    std::string fname;
};

Params getParams(int argc, char** argv) {
      int c;
      char *opt_val = NULL;
      Params result;
      while ((c = getopt (argc, argv, "a:b:r:c:f:")) != -1) {
          switch(c) {
            case 'a':
                opt_val = optarg;
                result.a = strtol(opt_val, NULL, 10);
                break;
            case 'b':
                opt_val = optarg;
                result.b = strtol(opt_val, NULL, 10);
                break;
            case 'r':
                opt_val = optarg;
                result.rows = strtol(opt_val, NULL, 10);
                break;
            case 'c':
                opt_val = optarg;
                result.cols = strtol(opt_val, NULL, 10);
                break;
            case 'f':
                opt_val = optarg;
                result.fname = std::string(opt_val);
                break;
          }
      }
      return result;
}


const double EPSILON = 0.0001;

int main(int argc, char **argv) {
    int size, rank;
    Params pars = getParams(argc, argv);
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD,&size);
    MPI_Comm_rank(MPI_COMM_WORLD,&rank);
    hipSetDevice(0);
    long A = pars.a, B = pars.b;
    int totalRows = pars.cols, totalCols = pars.rows;
    long rowsShift, colsShift;
    long rows, cols;
    MPI_Status status;
    double start;
    int sizePower = getPowOfTwo(size);
    PointUi ps = splitFunction(totalRows, totalCols, sizePower);
    int procRows =  1 << ps.first, procCols = 1<< ps.second;
    std::map<int, Mesh> splited;
    if (rank == 0) {
        start = MPI_Wtime();
        MeshConfig globalConf = {PointUi(0,0), PointUi(A,B), 0, 0, totalRows, totalCols};
        Mesh result(totalRows, totalCols, globalConf);
        initMeshBoundaries(result, phi);
        splited = splitMesh(result, sizePower);
        for(std::map<int, Mesh>::iterator itr = splited.begin(); itr != splited.end(); ++itr) {
            long r = itr->second.getRows();
            long c = itr->second.getColumns();
            long rShift = itr->second.getRowsShift();
            long cShift = itr->second.getColumnsShift();
            if(itr->first != rank) {
                MPI_Send(&r, 1, MPI_LONG, itr->first, 0, MPI_COMM_WORLD);
                MPI_Send(&c, 1, MPI_LONG, itr->first, 0, MPI_COMM_WORLD);
                MPI_Send(&rShift, 1, MPI_LONG, itr->first, 0, MPI_COMM_WORLD);
                MPI_Send(&cShift, 1, MPI_LONG, itr->first, 0, MPI_COMM_WORLD);
            } else {
                rows = r;
                cols = c;
                rowsShift = rShift;
                colsShift = cShift;
            }
        }
    } else {
        MPI_Recv(&rows, 1, MPI_LONG, 0, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
        MPI_Recv(&cols, 1, MPI_LONG, 0, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
        MPI_Recv(&rowsShift, 1, MPI_LONG, 0, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
        MPI_Recv(&colsShift, 1, MPI_LONG, 0, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
    }
    Mesh curMesh;
    if (rank == 0) {
        for(std::map<int, Mesh>::iterator itr = splited.begin(); itr != splited.end(); ++itr) {
            if(itr->first != rank) {
                MPI_Send(itr->second.getData(), itr->second.getRows()*itr->second.getColumns(),
                        MPI_DOUBLE, itr->first, 0, MPI_COMM_WORLD);
            }
        }
        curMesh = splited[0];
    } else {
        double *recdata = new double[rows*cols];
        MPI_Recv(recdata, rows*cols, MPI_DOUBLE, 0, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
        MeshConfig conf = { PointUi(0,0), PointUi(A,B),rowsShift, colsShift, totalRows, totalCols };
        curMesh = Mesh(rows, cols, recdata, conf);
    }
    int procCol = rank%procCols;
    int procRow = (rank - procCol) / procCols;

    long left = procCol - 1 >=0 ? procRow*procCols + procCol - 1 : -1;
    long right = procCol + 1 < procCols ? procRow*procCols + procCol + 1 : -1;
    long up = procRow - 1 >=0? (procRow - 1)*procCols + procCol : -1;
    long down = procRow + 1 < procRows ? (procRow + 1)*procCols + procCol : -1;

    MPI_Barrier(MPI_COMM_WORLD);
    dim3 gridDim;
    gridDim.x = (int)((curMesh.getRows() + 2) / BLOCK_SIZE_X + 1);
    gridDim.y = (int)((curMesh.getColumns() + 2) / BLOCK_SIZE_Y + 1);

    CudaIterator iter(gridDim, curMesh, rank, left, right, up, down, size);
    double err = iter.iterate();
    int iterCount = 1;
    while(err > EPSILON) {
        err = iter.iterate();
        if (rank == 0) {
            std::cout <<"Iteration: " << iterCount++ <<" Error: " << err <<"\n";
        }
    }
    iter.getPMesh(curMesh);
    if (rank != 0) {
        MPI_Send(&rows, 1, MPI_LONG, 0, 0, MPI_COMM_WORLD);
        MPI_Send(&cols, 1, MPI_LONG, 0, 0, MPI_COMM_WORLD);
        MPI_Send(&rowsShift, 1, MPI_LONG, 0, 0, MPI_COMM_WORLD);
        MPI_Send(&colsShift, 1, MPI_LONG, 0, 0, MPI_COMM_WORLD);
        MPI_Send(curMesh.getData(), rows*cols, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD);
    } else {
        std::map<int, Mesh> submeshs;
        submeshs[0] = curMesh;
        std::vector<MPI_Request> requests;
        for (int i = 1; i < size; ++i ) {
            MPI_Recv(&rows, 1, MPI_LONG, i, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
            MPI_Recv(&cols, 1, MPI_LONG, i, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
            MPI_Recv(&rowsShift, 1, MPI_LONG, i, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
            MPI_Recv(&colsShift, 1, MPI_LONG, i, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
            double *recIdata = new double[rows*cols];
            MPI_Recv(recIdata, rows*cols, MPI_DOUBLE, i, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
            MeshConfig conf = { PointUi(0,0), PointUi(A,B), rowsShift, colsShift,  totalRows, totalCols };
            Mesh curMesh(rows, cols, recIdata, conf);
            submeshs[i] = curMesh;
        }
        Mesh result = collectMesh(submeshs);
        double elapsed = MPI_Wtime() - start;
        std::ofstream ofs(pars.fname.c_str());
        dropToStream(ofs, result);
        ofs <<"stats:" <<iterCount <<'\t' <<totalRows << '\t' << totalCols << '\t' << elapsed;
    }

    MPI_Finalize();
    return 0;
}
